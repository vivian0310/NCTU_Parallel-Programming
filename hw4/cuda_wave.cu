/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
//void init_line(void);

void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
// void init_line(void)
// {
//    int i, j;
//    float x, fac, k, tmp;

//    /* Calculate initial values based on sine curve */
//    fac = 2.0 * PI;
//    k = 0.0; 
//    tmp = tpoints - 1;
//    for (j = 1; j <= tpoints; j++) {
//       x = k/tmp;
//       values[j] = sin (fac * x);
//       k = k + 1.0;
//    } 

//    /* Initialize old values array */
//    for (i = 1; i <= tpoints; i++) 
//       oldval[i] = values[i];
// }

// /**********************************************************************
//  *      Calculate new values using wave equation
//  *********************************************************************/
// void do_math(int i)
// {
//    float dtime, c, dx, tau, sqtau;

//    dtime = 0.3;
//    c = 1.0;
//    dx = 1.0;
//    tau = (c * dtime / dx);
//    sqtau = tau * tau;
//    newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
// }

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void update_parallel(float *values_D, int tpoints, int nsteps)
{
   // init line
   int i;
   float x, fac, tmp;
   int k;
   k = blockIdx.x * blockDim.x + (threadIdx.x + 1); // *value = 1 base
   /* Calculate initial values based on sine curve */
   float old_tmp, new_tmp, val_tmp;
   fac = 2.0 * PI;
   tmp = tpoints - 1;
  
   if (k <= tpoints){   
      //x = k/tmp;
      x = (k-1) / tmp;
      val_tmp = sin (fac * x);
      old_tmp = val_tmp;

      float dtime, c, dx, tau, sqtau;
      dtime = 0.3;
      c = 1.0;
      dx = 1.0;
      tau = (c * dtime / dx);
      sqtau = tau * tau;
  
      /* Update values for each time step */
      for (i = 1; i<= nsteps; i++) {
         /* Update points along line for this time step */
            /* global endpoints */
         if ((k == 1) || (k  == tpoints))
            new_tmp = 0.0;
         else
            // do math 
            new_tmp = (2.0 * val_tmp) - old_tmp + (sqtau * (-2.0) * val_tmp);
         
         /* Update old values with new values */
         old_tmp = val_tmp;
         val_tmp = new_tmp;
      }

      // read back from device
      values_D[k] = val_tmp;
   }
}


/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();

   // define cuda parameters
   float *values_D, *oldval_D, *newval_D;
   int size = (tpoints+2) * sizeof(float);
   hipMalloc((void**)&values_D, size);
   //cudaMalloc((void**)$oldval_D, size);
   //cudaMalloc((void**)$newval_D, size);
   //cudaMemcpy(values_D, values, size, cudaMemcpyHostToDevice);
   //cudaMemcpy(oldval_D, oldval, size, cudaMemcpyHostToDevice);
   //cudaMemcpy(newval_D, newval, size, cudaMemcpyHostToDevice);

   // dimension = 1D 
   int threadsPerBlock = 1024;
   int numBlocks = ((tpoints%threadsPerBlock) == 0)? \
                     (tpoints/threadsPerBlock) : (tpoints/threadsPerBlock) + 1;

	printf("Initializing points on the line...\n");
	//init_line();
	printf("Updating all points for all time steps...\n");
	//update();
	update_parallel <<<numBlocks, threadsPerBlock>>> (values_D, tpoints, nsteps);
   hipMemcpy(values, values_D, size, hipMemcpyDeviceToHost);

   printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	
   hipFree(values_D);
	return 0;
}